#include "hip/hip_runtime.h"
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Math.cuh>

namespace at { namespace native {

void acos_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "acos_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::acos(a);
    });
  });
}

void asin_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "asin_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::asin(a);
    });
  });
}

void atan_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "atan_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::atan(a);
    });
  });
}

void sin_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "sin_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::sin(a);
    });
  });
}

void cos_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.dtype(), "cos_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::cos(a);
    });
  });
}

void sinh_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "sinh_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::sinh(a);
    });
  });
}

void cosh_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "cosh_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::cosh(a);
    });
  });
}

void tanh_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.dtype(), "tanh_cuda", [&]() {
    AT_SKIP_BFLOAT16_IF_NOT_ROCM(scalar_t, "tanh_cuda", [&] {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return std::tanh(a);
      });
    });
  });
}

void tan_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "tan_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::tan(a);
    });
  });
}

REGISTER_DISPATCH(acos_stub, &acos_kernel_cuda);
REGISTER_DISPATCH(asin_stub, &asin_kernel_cuda);
REGISTER_DISPATCH(atan_stub, &atan_kernel_cuda);
REGISTER_DISPATCH(sin_stub, &sin_kernel_cuda);
REGISTER_DISPATCH(cos_stub, &cos_kernel_cuda);
REGISTER_DISPATCH(sinh_stub, &sinh_kernel_cuda);
REGISTER_DISPATCH(cosh_stub, &cosh_kernel_cuda);
REGISTER_DISPATCH(tanh_stub, &tanh_kernel_cuda);
REGISTER_DISPATCH(tan_stub, &tan_kernel_cuda);

}} // namespace at::native
